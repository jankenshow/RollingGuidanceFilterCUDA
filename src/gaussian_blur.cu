#include "hip/hip_runtime.h"
#include "gaussian_blur.h"
#include "kernels.h"
#include "utils.h"

namespace rgf {

__global__ void gaussian_blur_kernel(const float* input, float* output,
                                     int width, int height, float sigma) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= width || y >= height) return;

  int idx = y * width + x;
  float sum = 0.0f;
  float norm = 0.0f;
  int radius = int(ceilf(2.0f * sigma));

  for (int dy = -radius; dy <= radius; ++dy) {
    for (int dx = -radius; dx <= radius; ++dx) {
      int xx = clamp(x + dx, 0, width - 1);
      int yy = clamp(y + dy, 0, height - 1);
      int nidx = yy * width + xx;

      float weight = expf(-(dx * dx + dy * dy) / (2.0f * sigma * sigma));
      sum += weight * input[nidx];
      norm += weight;
    }
  }

  output[idx] = sum / norm;
}

__global__ void gaussian_blur_kernel_multi(const float* input, float* output,
                                           int width, int height, int channels,
                                           float sigma) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= width || y >= height) return;

  int radius = int(ceilf(2.0f * sigma));

  for (int c = 0; c < channels; ++c) {
    int idx = (y * width + x) * channels + c;
    float sum = 0.0f;
    float norm = 0.0f;

    for (int dy = -radius; dy <= radius; ++dy) {
      for (int dx = -radius; dx <= radius; ++dx) {
        int xx = clamp(x + dx, 0, width - 1);
        int yy = clamp(y + dy, 0, height - 1);
        int nidx = (yy * width + xx) * channels + c;

        float weight = expf(-(dx * dx + dy * dy) / (2.0f * sigma * sigma));
        sum += weight * input[nidx];
        norm += weight;
      }
    }

    output[idx] = sum / norm;
  }
}

void gaussian_blur_cuda(const float* input, float* output, int width,
                        int height, int channels, float sigma) {
  size_t bytes = width * height * channels * sizeof(float);
  float *d_input = nullptr, *d_output = nullptr;

  hipMalloc(&d_input, bytes);
  hipMalloc(&d_output, bytes);

  hipMemcpy(d_input, input, bytes, hipMemcpyHostToDevice);

  dim3 threads(16, 16);
  dim3 blocks((width + threads.x - 1) / threads.x,
              (height + threads.y - 1) / threads.y);

  if (channels == 1) {
    gaussian_blur_kernel<<<blocks, threads>>>(d_input, d_output, width, height,
                                              sigma);
  } else {
    gaussian_blur_kernel_multi<<<blocks, threads>>>(d_input, d_output, width,
                                                    height, channels, sigma);
  }

  hipMemcpy(output, d_output, bytes, hipMemcpyDeviceToHost);

  hipFree(d_input);
  hipFree(d_output);
}

}  // namespace rgf