#include "hip/hip_runtime.h"
#include "kernels.h"
#include "utils.h"
#include <iostream>

namespace rgf {

__global__ void rgf_bilateral_kernel(hipTextureObject_t texInput, hipTextureObject_t texGuide, hipSurfaceObject_t surfOutput, int width, int height, float sigma_s, float sigma_r) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;
    
    float center = uchar1_to_float(tex2D<uchar1>(texGuide, x, y));
    float norm = 0.0f, sum = 0.0f;
    int radius = int(ceilf(2.0f * sigma_s));
    
    for (int dy = -radius; dy <= radius; ++dy) {
        for (int dx = -radius; dx <= radius; ++dx) {
            if (x + dx < 0 || x + dx >= width || y + dy < 0 || y + dy >= height) continue;
                        
            float spatial = expf(-(dx*dx + dy*dy) / (2.0f * sigma_s * sigma_s));
            float guide_val = uchar1_to_float(tex2D<uchar1>(texGuide, x + dx, y + dy));
            float range = expf(-((guide_val - center) * (guide_val - center)) / (2.0f * sigma_r * sigma_r));
            float w = spatial * range;
            
            norm += w;
            sum += w * uchar1_to_float(tex2D<uchar1>(texInput, x + dx, y + dy));
        }
    }
    
    surf2Dwrite(float_to_uchar1(sum / norm), surfOutput, x * sizeof(uchar1), y);
}

__global__ void rgf_bilateral_kernel_multi(hipTextureObject_t texInput, hipTextureObject_t texGuide, hipSurfaceObject_t surfOutput, int width, int height, int channels, float sigma_s, float sigma_r) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;
    
    int radius = int(ceilf(2.0f * sigma_s));
    float4 center = uchar4_to_float4(tex2D<uchar4>(texGuide, x, y));
    float4 sum = make_float4(0.0f, 0.0f, 0.0f, 0.0f), norm = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    
    for (int dy = -radius; dy <= radius; ++dy) {
        for (int dx = -radius; dx <= radius; ++dx) {
            if (x + dx < 0 || x + dx >= width || y + dy < 0 || y + dy >= height) continue;

            float4 spatial;
            spatial.x = expf(-(dx*dx + dy*dy) / (2.0f * sigma_s * sigma_s));
            spatial.y = expf(-(dx*dx + dy*dy) / (2.0f * sigma_s * sigma_s));
            spatial.z = expf(-(dx*dx + dy*dy) / (2.0f * sigma_s * sigma_s));
            spatial.w = expf(-(dx*dx + dy*dy) / (2.0f * sigma_s * sigma_s));
            float4 guide_val = uchar4_to_float4(tex2D<uchar4>(texGuide, x + dx, y + dy));
            float4 range;
            range.x = expf(-((guide_val.x - center.x) * (guide_val.x - center.x)) / (2.0f * sigma_r * sigma_r));
            range.y = expf(-((guide_val.y - center.y) * (guide_val.y - center.y)) / (2.0f * sigma_r * sigma_r));
            range.z = expf(-((guide_val.z - center.z) * (guide_val.z - center.z)) / (2.0f * sigma_r * sigma_r));
            float4 w = spatial * range;
            
            norm += w;
            sum += w * uchar4_to_float4(tex2D<uchar4>(texInput, x + dx, y + dy));
        }
    }
    
    surf2Dwrite(float4_to_uchar4(sum / norm), surfOutput, x*sizeof(uchar4), y);
}

void rolling_guidance_filter_cuda(const unsigned char* input, unsigned char* output, int width, int height, int channels, float sigma_s, float sigma_r, int iterations) {
    int orig_channels = channels;
    
    unsigned char *input_tmp, *output_tmp;
    hipArray_t d_inputArray, d_guideArray, d_outputArray;
    hipChannelFormatDesc channelDesc;
    if (channels == 1) {
        channelDesc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
    } else if (channels == 3) {
        channels = 4;
        channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
        input_tmp = new unsigned char[width * height * 4];
        for (int i = 0; i < width * height; i++) {
            input_tmp[i * 4] = input[i * 3];
            input_tmp[i * 4 + 1] = input[i * 3 + 1];
            input_tmp[i * 4 + 2] = input[i * 3 + 2];
            input_tmp[i * 4 + 3] = 0;
        }
    } else if (channels == 4) {
        channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
    } else {
        std::cout << "Invalid number of channels: %d\n" << std::endl;
        return;
    }
    CHECK(hipMallocArray(&d_inputArray, &channelDesc, width, height));
    CHECK(hipMallocArray(&d_guideArray, &channelDesc, width, height));
    CHECK(hipMallocArray(&d_outputArray, &channelDesc, width, height, hipArraySurfaceLoadStore));
    if (orig_channels == 3) {
        CHECK(hipMemcpy2DToArray(d_inputArray, 0, 0, input_tmp, width * channels * sizeof(unsigned char), width * channels * sizeof(unsigned char), height, hipMemcpyHostToDevice));
    } else {
        CHECK(hipMemcpy2DToArray(d_inputArray, 0, 0, input, width * channels * sizeof(unsigned char), width * channels * sizeof(unsigned char), height, hipMemcpyHostToDevice));
    }
    
    // テクスチャメモリの宣言
    hipTextureObject_t texInput;
    hipTextureObject_t texGuide;
    hipSurfaceObject_t surfOutput;
    
    // テクスチャの設定
    hipTextureDesc texDescr = {};
    texDescr.normalizedCoords = false;
    texDescr.filterMode = hipFilterModePoint;
    texDescr.addressMode[0] = hipAddressModeWrap;
    texDescr.addressMode[1] = hipAddressModeWrap;
    texDescr.addressMode[2] = hipAddressModeWrap;
    texDescr.readMode = hipReadModeElementType;

    hipResourceDesc texRes = {};
    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = d_inputArray;
    CHECK(hipCreateTextureObject(&texInput, &texRes, &texDescr, nullptr));
    
    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = d_guideArray;
    CHECK(hipCreateTextureObject(&texGuide, &texRes, &texDescr, nullptr));

    hipResourceDesc surfRes = {};
    surfRes.resType = hipResourceTypeArray;
    surfRes.res.array.array = d_outputArray;
    CHECK(hipCreateSurfaceObject(&surfOutput, &surfRes));
    
    dim3 threads(16, 16);
    dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);
    
    // ガウシアンブラーを実行
    if (channels == 1) {
        gaussian_blur_kernel<<<blocks, threads>>>(texInput, surfOutput, width, height, sigma_s);
    } else {
        gaussian_blur_kernel_multi<<<blocks, threads>>>(texInput, surfOutput, width, height, channels, sigma_s);
    }
    CHECK(hipDeviceSynchronize());
    
    // イテレーション実行
    for (int i = 1; i < iterations; ++i) {
        CHECK(hipMemcpy2DArrayToArray(d_guideArray, 0, 0, d_outputArray, 0, 0, width * channels * sizeof(unsigned char), height, hipMemcpyDeviceToDevice));

        if (channels == 1) {
            rgf_bilateral_kernel<<<blocks, threads>>>(texInput, texGuide, surfOutput, width, height, sigma_s, sigma_r);
        } else {
            rgf_bilateral_kernel_multi<<<blocks, threads>>>(texInput, texGuide, surfOutput, width, height, channels, sigma_s, sigma_r);
        }
        CHECK(hipDeviceSynchronize());
        // std::swap(d_guideArray, d_outputArray);
        // CHECK(hipCreateTextureObject(&texGuide, &texRes, &texDescr, nullptr));
        // CHECK(hipCreateSurfaceObject(&surfOutput, &surfRes));
    }
    
    // 結果をホストメモリにコピー
    if (orig_channels == 3) {
        output_tmp = new unsigned char[width * height * channels];
        CHECK(hipMemcpy2DFromArray(output_tmp, width * channels * sizeof(unsigned char), d_outputArray, 0, 0, width * channels * sizeof(unsigned char), height, hipMemcpyDeviceToHost));
        for (int i = 0; i < width * height; i++) {
            output[i * 3] = output_tmp[i * 4];
            output[i * 3 + 1] = output_tmp[i * 4 + 1];
            output[i * 3 + 2] = output_tmp[i * 4 + 2];
        }
        delete[] input_tmp;
        delete[] output_tmp;
    } else {
        CHECK(hipMemcpy2DFromArray(output, width * channels * sizeof(unsigned char), d_outputArray, 0, 0, width * channels * sizeof(unsigned char), height, hipMemcpyDeviceToHost));
    }
    
    // リソースの解放
    CHECK(hipDestroyTextureObject(texInput));
    CHECK(hipDestroyTextureObject(texGuide));
    CHECK(hipDestroySurfaceObject(surfOutput));
    CHECK(hipFreeArray(d_inputArray));
    CHECK(hipFreeArray(d_guideArray));
    CHECK(hipFreeArray(d_outputArray));
}
} // namespace rgf
